#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include "gpu_nms.hpp"
#include <vector>
#include <iostream>

#define CUDA_CHECK(condition) \
    /* Code block avoids redefinition of hipError_t error */ \
    do { \
        hipError_t error = condition; \
        if (error != hipSuccess) { \
            std::cout << hipGetErrorString(error) << std::endl; \
        } \
    } while (0)

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
    float left = max();
}